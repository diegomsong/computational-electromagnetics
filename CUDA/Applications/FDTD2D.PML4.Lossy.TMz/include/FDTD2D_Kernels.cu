
#include <hip/hip_runtime.h>
template <unsigned int BlockX, unsigned int BlockY> __global__ void FDTD2DKernel(
							float *Hx,
							float *Bx,
							float *Hy,
							float *By,
							float *Ez,
							float *Dz,
							float *Dzx,
							float *Dzy,
							float *urHx,
							float *urHy,
							float *erEz,
							float *ScmHx,
							float *ScmHy,
							float *Sc,
							const float delta,
							const float dtscalar,
							const float dt,
							const uint PMLw,
							const float e0,
							const float u0,
							const float Two_pi_f_deltat,
							const uint NHW,
							const uint Is,
							const uint Js,
							const uint IHx,
							const uint JHx,
							const uint IHy,
							const uint JHy,
							const uint IEz,
							const uint JEz,
							const uint n,
							const uint n0,
							const uint n1,
							const uint n2,
							const uint flag)
{
    uint i = BlockX*blockIdx.x+threadIdx.x;
	uint j = BlockY*blockIdx.y+threadIdx.y;

	// Half time step flag is either 0 or 1 indicating whether magnetic field or electric field is to be calculated, respectively.
	if (flag == 0)
	{
		if (i < IHx && j < JHx)
		{
			Bx[i+IHx*j+IHx*JHx*n2] = (1-ScmHx[i+IHx*j])/(1+ScmHx[i+IHx*j]) * Bx[i+IHx*j+IHx*JHx*n1] + ( (dt/delta)/(1+ScmHx[i+IHx*j]) * (Ez[i+IEz*j+IEz*JEz*n1]-Ez[i+IEz*(j+1)+IEz*JEz*n1]) );
			Hx[i+IHx*j+IHx*JHx*n2] = Bx[i+IHx*j+IHx*JHx*n2]/(u0*urHx[i+IHx*j]);

			By[(i+1)+IHy*(j+1)+IHy*JHy*n2] = (1-ScmHy[(i+1)+IHy*(j+1)])/(1+ScmHy[(i+1)+IHy*(j+1)]) * By[(i+1)+IHy*(j+1)+IHy*JHy*n1] + ( (dt/delta)/(1+ScmHy[(i+1)+IHy*(j+1)]) * (Ez[(i+1)+IEz*(j+1)+IEz*JEz*n1]-Ez[i+IEz*(j+1)+IEz*JEz*n1]) );
			Hy[(i+1)+IHy*(j+1)+IHy*JHy*n2] = By[(i+1)+IHy*(j+1)+IHy*JHy*n2]/(u0*urHy[(i+1)+IHy*(j+1)]);
		}
	}
	else

	{
		if (i < IEz && j != 0 && j < JEz-1 )
		{
			Dz[i+IEz*j+IEz*JEz*n2] = (1-Sc[i+IEz*j])/(1+Sc[i+IEz*j]) * Dz[i+IEz*j+IEz*JEz*n1] + ( (dt/delta)/(1+Sc[i+IEz*j]) * ( Hy[(i+1)+IHy*j+IHy*JHy*n2] - Hy[i+IHy*j+IHy*JHy*n2] - Hx[i+IHx*j+IHx*JHx*n2] + Hx[i+IHx*(j-1)+IHx*JHx*n2]) );
			Ez[i+IEz*j+IEz*JEz*n2] = Dz[i+IEz*j+IEz*JEz*n2]/(e0*erEz[i+IEz*j]);

			// Source.
			if (j == Js && n < NHW)
			{
				Ez[i+IEz*j+IEz*JEz*n2] = Ez[i+IEz*j+IEz*JEz*n2] + 1 * sin (Two_pi_f_deltat * n) / dtscalar;
				Dz[i+IEz*j+IEz*JEz*n2] = e0 * Ez[i+IEz*j+IEz*JEz*n2];
			}
		}
	}
}
