#include "hip/hip_runtime.h"
// Field arrays indexing macros.
#define Ez(i,j,n) Ez_[(i)+IEz*(j)+IEz*JEz*(n)]
#define Dz(i,j,n) Dz_[(i)+IEz*(j)+IEz*JEz*(n)]
#define EzMask(i,j) EzMask_[(i)+IEz*(j)]
#define Hx(i,j,n) Hx_[(i)+IHx*(j)+IHx*JHx*(n)]
#define Bx(i,j,n) Bx_[(i)+IHx*(j)+IHx*JHx*(n)]
#define BxAve(i,j,n) BxAve_[(i)+IHy*(j)+IHy*JHy*(n)]
#define Hy(i,j,n) Hy_[(i)+IHy*(j)+IHy*JHy*(n)]
#define By(i,j,n) By_[(i)+IHy*(j)+IHy*JHy*(n)]
#define ByAve(i,j,n) ByAve_[(i)+IHx*(j)+IHx*JHx*(n)]
// Auxiliary scalar arrays indexing macros.
#define ax(i,j,k) ax_[(i)+IHx*(j)+IHx*JHx*(k)]
#define ay(i,j,k) ay_[(i)+IHy*(j)+IHy*JHy*(k)]
#define az(i,j,k) az_[(i)+IEz*(j)+IEz*JEz*(k)]
// PML arrays indexing macros.
#define PsiEzX(i,j) PsiEzX_[(i)+IEz*(j)]
#define PsiEzY(i,j) PsiEzY_[(i)+IEz*(j)]
#define PsiHyX(i,j) PsiHyX_[(i)+IHy*(j)]
#define PsiHxY(i,j) PsiHxY_[(i)+IHx*(j)]

#include <FDTD2DCloak.hpp>

// Dry run kernel.
template <unsigned int BlockX, unsigned int BlockY> __global__ void FDTD2DCloakKernel_DryRun_M(
							const unsigned int I,
							const unsigned int J,
							const unsigned int PMLw,
							const unsigned int PulseWidth,
							const unsigned int td,
							const unsigned int SourceChoice,
							const unsigned int SourcePlane,
							const unsigned int SourceLocationX,
							const unsigned int SourceLocationY,
							const PRECISION c,
							const PRECISION pi,
							const PRECISION e0,
							const PRECISION u0,
							const PRECISION dt,
							const PRECISION delta,
							const PRECISION Sc,
							const PRECISION f,
							const PRECISION fp,
							const PRECISION dr,
							const unsigned int IEz, const unsigned int JEz,
							const unsigned int IHx, const unsigned int JHx,
							const unsigned int IHy, const unsigned int JHy,
							const PRECISION *Ez_, const PRECISION *Dz_, PRECISION *Hx_, PRECISION *Bx_, PRECISION *Hy_, PRECISION *By_,
							const PRECISION *PsiEzX_, const PRECISION *PsiEzY_, PRECISION *PsiHyX_, PRECISION *PsiHxY_,
							const PRECISION kappex, const PRECISION kappey, const PRECISION kappmx, const PRECISION kappmy,
							const PRECISION bex, const PRECISION bey, const PRECISION bmx, const PRECISION bmy,
							const PRECISION Cex, const PRECISION Cey, const PRECISION Cmx, const PRECISION Cmy,
							PRECISION *Ezi,
							const unsigned int x1,
							const unsigned int n,
							const unsigned int np,
							const unsigned int n0,
							const unsigned int nf)
{
	const unsigned int i = BlockX*blockIdx.x+threadIdx.x;
	const unsigned int j = BlockY*blockIdx.y+threadIdx.y;

	// Calculation of PsiHxY.
	if (j>0 && j<JHx-1)
		PsiHxY(i,j) = (Cmy/delta)*(-Ez(i,j,n0) + Ez(i,j-1,n0)) + bmy*PsiHxY(i,j);

	// Bx in normal space.
	if( j>PMLw && j<JHx-PMLw-1)
	{
		Bx(i,j,nf) = Bx(i,j,n0) + (-Ez(i,j,n0) + Ez(i,j-1,n0)) * dt/delta;
		Hx(i,j,nf) = Bx(i,j,nf)/u0;
	}
	// Bx in lower PML.
	if (j>0 && j<PMLw+1)
	{
		Bx(i,j,nf) = Bx(i,j,n0) + dt*((1./kappmy)*(-Ez(i,j,n0) + Ez(i,j-1,n0)) * 1./delta + PsiHxY(i,j));
		Hx(i,j,nf) = Bx(i,j,nf)/u0;
	}
	// Bx in upper PML.
	if (j>JHx-PMLw-2 && j<JHx-1)
	{
		Bx(i,j,nf) = Bx(i,j,n0) + dt*((1./kappmy)*(-Ez(i,j,n0) + Ez(i,j-1,n0)) * 1./delta + PsiHxY(i,j));
		Hx(i,j,nf) = Bx(i,j,nf)/u0;
	}

	// ========================== By and Hy ==========================
	if (i<IHy-1)
	{
		// PsiHyX array.
		if (j<JHy)
			PsiHyX(i,j) = (Cmx/delta)*(Ez(i+1,j,n0)-Ez(i,j,n0)) + bmx*PsiHyX(i,j);

		// By in normal space.
		if (j>PMLw-1 && j<JHy-PMLw)
		{
			By(i,j,nf) = By(i,j,n0) + (Ez(i+1,j,n0) - Ez(i,j,n0)) * dt/delta;
			Hy(i,j,nf) = By(i,j,nf)/u0;
		}
		// By in Lower PML.
		if (j<PMLw)
		{
			By(i,j,nf) = By(i,j,n0) + dt*((1./kappmx)*(Ez(i+1,j,n0) - Ez(i,j,n0)) * 1./delta + PsiHyX(i,j));
			Hy(i,j,nf) = By(i,j,nf)/u0;
		}
		// By in upper PML.
		if (j>JHy-PMLw-1 && j<JHy)
		{
			By(i,j,nf) = By(i,j,n0) + dt*((1./kappmx)*(Ez(i+1,j,n0) - Ez(i,j,n0)) * 1./delta + PsiHyX(i,j));
			Hy(i,j,nf) = By(i,j,nf)/u0;
		}
	}
	else
	{
		// PsiHyX array.
		if (j<JHy)
			PsiHyX(IHy-1,j) = (Cmx/delta)*(Ez(0,j,n0)-Ez(IHy-1,j,n0)) + bmx*PsiHyX(IHy-1,j); // PBC

		// By in normal space.
		if (j>PMLw-1 && j<JHy-PMLw)
		{
			By(IHy-1,j,nf) = By(IHy-1,j,n0) + (Ez(0,j,n0) - Ez(IHy-1,j,n0)) * dt/delta; // PBC
			Hy(IHy-1,j,nf) = By(IHy-1,j,nf)/u0; // PBC
		}
		// By in Lower PML.
		if (j<PMLw)
		{
			By(IHy-1,j,nf) = By(IHy-1,j,n0) + dt*((1./kappmx)*(Ez(0,j,n0) - Ez(IHy-1,j,n0)) * 1./delta + PsiHyX(IHy-1,j)); // PBC
			Hy(IHy-1,j,nf) = By(IHy-1,j,nf)/u0; // PBC
		}
		// By in upper PML.
		if (j>JHy-PMLw-1 && j<JHy)
		{
			By(IHy-1,j,nf) = By(IHy-1,j,n0) + dt*((1./kappmx)*(Ez(0,j,n0) - Ez(IHy-1,j,n0)) * 1./delta + PsiHyX(IHy-1,j)); // PBC
			Hy(IHy-1,j,nf) = By(IHy-1,j,nf)/u0; // PBC
		}
	}
}
template <unsigned int BlockX, unsigned int BlockY> __global__ void FDTD2DCloakKernel_DryRun_E(
							const unsigned int I,
							const unsigned int J,
							const unsigned int PMLw,
							const unsigned int PulseWidth,
							const unsigned int td,
							const unsigned int SourceChoice,
							const unsigned int SourcePlane,
							const unsigned int SourceLocationX,
							const unsigned int SourceLocationY,
							const PRECISION c,
							const PRECISION pi,
							const PRECISION e0,
							const PRECISION u0,
							const PRECISION dt,
							const PRECISION delta,
							const PRECISION Sc,
							const PRECISION f,
							const PRECISION fp,
							const PRECISION dr,
							const unsigned int IEz, const unsigned int JEz,
							const unsigned int IHx, const unsigned int JHx,
							const unsigned int IHy, const unsigned int JHy,
							PRECISION *Ez_, PRECISION *Dz_, const PRECISION *Hx_, const PRECISION *Bx_, const PRECISION *Hy_, const PRECISION *By_,
							PRECISION *PsiEzX_, PRECISION *PsiEzY_, const PRECISION *PsiHyX_, const PRECISION *PsiHxY_,
							const PRECISION kappex, const PRECISION kappey, const PRECISION kappmx, const PRECISION kappmy,
							const PRECISION bex, const PRECISION bey, const PRECISION bmx, const PRECISION bmy,
							const PRECISION Cex, const PRECISION Cey, const PRECISION Cmx, const PRECISION Cmy,
							PRECISION *Ezi,
							const unsigned int x1,
							const unsigned int n,
							const unsigned int np,
							const unsigned int n0,
							const unsigned int nf)
{
	const unsigned int i = BlockX*blockIdx.x+threadIdx.x;
	const unsigned int j = BlockY*blockIdx.y+threadIdx.y;

	// ========================== Dz and Ez ==========================
	if (i>0)
	{
		// Psi arrays.
		if (j<JEz)
		{
			PsiEzX(i,j) = (Cex/delta)*(Hy(i,j,nf)-Hy(i-1,j,nf)) + bex*PsiEzX(i,j);
			PsiEzY(i,j) = (Cey/delta)*(-Hx(i,j+1,nf)+Hx(i,j,nf)) + bey*PsiEzY(i,j);
		}
		// Dz in normal space.
		if (j>PMLw-1 && j<JEz-PMLw)
		{
			Dz(i,j,nf) = Dz(i,j,n0) + (Hy(i,j,nf)-Hy(i-1,j,nf)-Hx(i,j+1,nf)+Hx(i,j,nf)) * dt/delta;
			Ez(i,j,nf) = Dz(i,j,nf)/e0;
		}
		// Dz in lower PML.
		if (j<PMLw)
		{
			Dz(i,j,nf) = Dz(i,j,n0) + dt*(((1./kappex)*(Hy(i,j,nf)-Hy(i-1,j,nf))+(1./kappey)*(-Hx(i,j+1,nf)+Hx(i,j,nf))) * 1./delta + PsiEzX(i,j) + PsiEzY(i,j));
			Ez(i,j,nf) = Dz(i,j,nf)/e0;
		}
		// Dz in upper PML.
		if (j>JEz-PMLw-1 && j<JEz)
		{
			Dz(i,j,nf) = Dz(i,j,n0) + dt*(((1./kappex)*(Hy(i,j,nf)-Hy(i-1,j,nf))+(1./kappey)*(-Hx(i,j+1,nf)+Hx(i,j,nf))) * 1./delta + PsiEzX(i,j) + PsiEzY(i,j));
			Ez(i,j,nf) = Dz(i,j,nf)/e0;
		}
	}
	else
	{
		// Psi arrays.
		if (j<JEz)
		{
			PsiEzX(0,j) = (Cex/delta)*(Hy(0,j,nf)-Hy(IEz-1,j,nf)) + bex*PsiEzX(0,j); // PBC
			PsiEzY(0,j) = (Cey/delta)*(-Hx(0,j+1,nf)+Hx(0,j,nf)) + bey*PsiEzY(0,j); // PBC
		}
		// Dz in normal space.
		if (j>PMLw-1 && j<JEz-PMLw)
		{
			Dz(0,j,nf) = Dz(0,j,n0) + (Hy(0,j,nf)-Hy(IEz-1,j,nf)-Hx(0,j+1,nf)+Hx(0,j,nf)) * dt/delta; // PBC
			Ez(0,j,nf) = Dz(0,j,nf)/e0; // PBC
		}
		// Dz in lower PML.
		if (j<PMLw)
		{
			Dz(0,j,nf) = Dz(0,j,n0) + dt*(((1./kappex)*(Hy(0,j,nf)-Hy(IEz-1,j,nf))+(1./kappey)*(-Hx(0,j+1,nf)+Hx(0,j,nf))) * 1./delta + PsiEzX(0,j) + PsiEzY(0,j)); // PBC
			Ez(0,j,nf) = Dz(0,j,nf)/e0; // PBC
		}
		// Dz in upper PML.
		if (j>JEz-PMLw-1 && j<JEz)
		{
			Dz(0,j,nf) = Dz(0,j,n0) + dt*(((1./kappex)*(Hy(0,j,nf)-Hy(IEz-1,j,nf))+(1./kappey)*(-Hx(0,j+1,nf)+Hx(0,j,nf))) * 1./delta + PsiEzX(0,j) + PsiEzY(0,j)); // PBC
			Ez(0,j,nf) = Dz(0,j,nf)/e0; // PBC
		}
	}

	// ====================== Source ======================
	if (SourcePlane == 1 && i<IEz && j == SourceLocationY)
	{
		if (SourceChoice == 1)
			Ez(i,j,nf) += exp(-1.*pow(((PRECISION)n-(PRECISION)td)/((PRECISION)PulseWidth/4.),2)) * Sc;
		else if (SourceChoice == 2)
			Ez(i,j,nf) += sin(2.*pi*f*(PRECISION)n*dt) * Sc;
		else if (SourceChoice == 3)
			Ez(i,j,nf) += (1.-2.*pow(pi*fp*((PRECISION)n*dt-dr),2))*exp(-1.*pow(pi*fp*((PRECISION)n*dt-dr),2)) * Sc;

		Dz(i,j,nf) = e0*Ez(i,j,nf);
	}
	else if (i == SourceLocationX && j == SourceLocationY)
	{
		if (SourceChoice == 1)
			Ez(i,j,nf) += exp(-1.*pow(((PRECISION)n-(PRECISION)td)/((PRECISION)PulseWidth/4.),2)) * Sc;
		else if (SourceChoice == 2)
			Ez(i,j,nf) += sin(2.*pi*f*(PRECISION)n*dt) * Sc;
		else if (SourceChoice == 3)
			Ez(i,j,nf) += (1.-2.*pow(pi*fp*((PRECISION)n*dt-dr),2))*exp(-1.*pow(pi*fp*((PRECISION)n*dt-dr),2)) * Sc;

		Dz(i,j,nf) = e0*Ez(i,j,nf);
	}
	if (j==x1)
		Ezi[n] = Ez(IEz/2,x1,nf); // Incident field.
}
// Simulation kernel.
template <unsigned int BlockX, unsigned int BlockY> __global__ void FDTD2DCloakKernel_Simulation_B(
							const unsigned int I,
							const unsigned int J,
							const unsigned int PMLw,
							const unsigned int PulseWidth,
							const unsigned int td,
							const unsigned int SourceChoice,
							const unsigned int SourcePlane,
							const unsigned int SourceLocationX,
							const unsigned int SourceLocationY,
							const PRECISION c,
							const PRECISION pi,
							const PRECISION e0,
							const PRECISION u0,
							const PRECISION dt,
							const PRECISION delta,
							const PRECISION Sc,
							const PRECISION f,
							const PRECISION fp,
							const PRECISION dr,
							const unsigned int IEz, const unsigned int JEz,
							const unsigned int IHx, const unsigned int JHx,
							const unsigned int IHy, const unsigned int JHy,
							const PRECISION *Ez_, const PRECISION *Dz_, PRECISION *Hx_, PRECISION *Bx_, PRECISION *Hy_, PRECISION *By_,
							const PRECISION *ax_, const PRECISION *ay_, const PRECISION *az_,
							const PRECISION *PsiEzX_, const PRECISION *PsiEzY_, PRECISION *PsiHyX_, PRECISION *PsiHxY_,
							const PRECISION kappex, const PRECISION kappey, const PRECISION kappmx, const PRECISION kappmy,
							const PRECISION bex, const PRECISION bey, const PRECISION bmx, const PRECISION bmy,
							const PRECISION Cex, const PRECISION Cey, const PRECISION Cmx, const PRECISION Cmy,
							PRECISION *Ezt, PRECISION *Eztt, PRECISION *Ezy1, PRECISION *Ezy2,
							const unsigned int x1, const unsigned int Y1, const unsigned int Y2,
							const unsigned int n,
							const unsigned int np,
							const unsigned int n0,
							const unsigned int nf)
{
	const unsigned int i = BlockX*blockIdx.x+threadIdx.x;
	const unsigned int j = BlockY*blockIdx.y+threadIdx.y;

	// ========================== Bx ==========================
	// Calculation of PsiHxY.
	if (j>0 && j<JHx-1)
		PsiHxY(i,j) = (Cmy/delta)*(-Ez(i,j,n0) + Ez(i,j-1,n0)) + bmy*PsiHxY(i,j);

	// Bx in normal space.
	if (j>PMLw && j<JHx-PMLw-1)
		Bx(i,j,nf) = Bx(i,j,n0) + (-Ez(i,j,n0) + Ez(i,j-1,n0)) * dt/delta;

		// Bx in lower PML.
	if (j>0 && j<PMLw+1)
		Bx(i,j,nf) = Bx(i,j,n0) + dt*((1./kappmy)*(-Ez(i,j,n0) + Ez(i,j-1,n0)) * 1./delta + PsiHxY(i,j));

	// Bx in upper PML.
	if (j>JHx-PMLw-2 && j<JHx-1)
		Bx(i,j,nf) = Bx(i,j,n0) + dt*((1./kappmy)*(-Ez(i,j,n0) + Ez(i,j-1,n0)) * 1./delta + PsiHxY(i,j));

	// ========================== By ==========================
	if (i<IHy-1)
	{
		// PsiHyX array.
		if (j<JHy)
			PsiHyX(i,j) = (Cmx/delta)*(Ez(i+1,j,n0)-Ez(i,j,n0)) + bmx*PsiHyX(i,j);

		// By in normal space.
		if (j>PMLw-1 && j<JHy-PMLw)
			By(i,j,nf) = By(i,j,n0) + (Ez(i+1,j,n0) - Ez(i,j,n0)) * dt/delta;

			// By in Lower PML.
		if (j<PMLw)
			By(i,j,nf) = By(i,j,n0) + dt*((1./kappmx)*(Ez(i+1,j,n0) - Ez(i,j,n0)) * 1./delta + PsiHyX(i,j));

		// By in upper PML.
		if (j>JHy-PMLw-1 && j<JHy)
			By(i,j,nf) = By(i,j,n0) + dt*((1./kappmx)*(Ez(i+1,j,n0) - Ez(i,j,n0)) * 1./delta + PsiHyX(i,j));
	}
	else
	{
		// PsiHyX array.
		if (j<JHy)
			PsiHyX(IHy-1,j) = (Cmx/delta)*(Ez(0,j,n0)-Ez(IHy-1,j,n0)) + bmx*PsiHyX(IHy-1,j); // PBC

		// By in normal space.
		if (j>PMLw-1 && j<JHy-PMLw)
			By(IHy-1,j,nf) = By(IHy-1,j,n0) + (Ez(0,j,n0) - Ez(IHy-1,j,n0)) * dt/delta; // PBC

		// By in Lower PML.
		if (j<PMLw)
			By(IHy-1,j,nf) = By(IHy-1,j,n0) + dt*((1./kappmx)*(Ez(0,j,n0) - Ez(IHy-1,j,n0)) * 1./delta + PsiHyX(IHy-1,j)); // PBC

		// By in upper PML.
		if (j>JHy-PMLw-1 && j<JHy)
			By(IHy-1,j,nf) = By(IHy-1,j,n0) + dt*((1./kappmx)*(Ez(0,j,n0) - Ez(IHy-1,j,n0)) * 1./delta + PsiHyX(IHy-1,j)); // PBC
	}
}
							// Simulation kernel.
template <unsigned int BlockX, unsigned int BlockY> __global__ void FDTD2DCloakKernel_Simulation_H(
							const unsigned int I,
							const unsigned int J,
							const unsigned int PMLw,
							const unsigned int PulseWidth,
							const unsigned int td,
							const unsigned int SourceChoice,
							const unsigned int SourcePlane,
							const unsigned int SourceLocationX,
							const unsigned int SourceLocationY,
							const PRECISION c,
							const PRECISION pi,
							const PRECISION e0,
							const PRECISION u0,
							const PRECISION dt,
							const PRECISION delta,
							const PRECISION Sc,
							const PRECISION f,
							const PRECISION fp,
							const PRECISION dr,
							const unsigned int IEz, const unsigned int JEz,
							const unsigned int IHx, const unsigned int JHx,
							const unsigned int IHy, const unsigned int JHy,
							const PRECISION *Ez_, const PRECISION *Dz_, PRECISION *Hx_, const PRECISION *Bx_, PRECISION *BxAve_, PRECISION *Hy_, const PRECISION *By_, PRECISION *ByAve_,
							const PRECISION *ax_, const PRECISION *ay_, const PRECISION *az_,
							const PRECISION *PsiEzX_, const PRECISION *PsiEzY_, PRECISION *PsiHyX_, PRECISION *PsiHxY_,
							const PRECISION kappex, const PRECISION kappey, const PRECISION kappmx, const PRECISION kappmy,
							const PRECISION bex, const PRECISION bey, const PRECISION bmx, const PRECISION bmy,
							const PRECISION Cex, const PRECISION Cey, const PRECISION Cmx, const PRECISION Cmy,
							PRECISION *Ezt, PRECISION *Eztt, PRECISION *Ezy1, PRECISION *Ezy2,
							const unsigned int x1, const unsigned int Y1, const unsigned int Y2,
							const unsigned int n,
							const unsigned int np,
							const unsigned int n0,
							const unsigned int nf)
{
	const unsigned int i = BlockX*blockIdx.x+threadIdx.x;
	const unsigned int j = BlockY*blockIdx.y+threadIdx.y;

	// ========================== BxAve and ByAve ==========================
	if (i>1 && i<IHy-2 && j>2+PMLw && j<JHx-PMLw-3)
	{
		BxAve(i,j,nf) = (Bx(i,j,nf)+Bx(i+1,j,nf)+Bx(i,j+1,nf)+Bx(i+1,j+1,nf))/4.;
		ByAve(i,j,nf) = (By(i,j,nf)+By(i-1,j,nf)+By(i,j-1,nf)+By(i-1,j-1,nf))/4.;
	}

	// ========================== Hx ==========================
	// Hx in normal space.
	if (j>PMLw && j<JHx-PMLw-1)
		Hx(i,j,nf) = (ax(i,j,0)*Bx(i,j,nf)+ax(i,j,1)*Bx(i,j,n0)+ax(i,j,2)*Bx(i,j,np)+ax(i,j,3)*ByAve(i,j,nf)+ax(i,j,4)*ByAve(i,j,n0)+ax(i,j,5)*ByAve(i,j,np)-ax(i,j,6)*Hx(i,j,n0)-ax(i,j,7)*Hx(i,j,np))/ax(i,j,8);

	// Bx in lower PML.
	if (j>0 && j<PMLw+1)
		Hx(i,j,nf) = Bx(i,j,nf)/u0;

	// Bx in upper PML.
	if (j>JHx-PMLw-2 && j<JHx-1)
		Hx(i,j,nf) = Bx(i,j,nf)/u0;

	// ========================== Hy ==========================
	// Hy in normal space.
	if (j>PMLw-1 && j<JHy-PMLw)
		Hy(i,j,nf) = (ay(i,j,0)*By(i,j,nf)+ay(i,j,1)*By(i,j,n0)+ay(i,j,2)*By(i,j,np)+ay(i,j,3)*BxAve(i,j,nf)+ay(i,j,4)*BxAve(i,j,n0)+ay(i,j,5)*BxAve(i,j,np)-ay(i,j,6)*Hy(i,j,n0)-ay(i,j,7)*Hy(i,j,np))/ay(i,j,8);

	// Hy in Lower PML.
	if (j<PMLw)
		Hy(i,j,nf) = By(i,j,nf)/u0;

	// Hy in upper PML.
	if (j>JHy-PMLw-1 && j<JHy)
		Hy(i,j,nf) = By(i,j,nf)/u0;
}
template <unsigned int BlockX, unsigned int BlockY> __global__ void FDTD2DCloakKernel_Simulation_E(
							const unsigned int I,
							const unsigned int J,
							const unsigned int PMLw,
							const unsigned int PulseWidth,
							const unsigned int td,
							const unsigned int SourceChoice,
							const unsigned int SourcePlane,
							const unsigned int SourceLocationX,
							const unsigned int SourceLocationY,
							const PRECISION c,
							const PRECISION pi,
							const PRECISION e0,
							const PRECISION u0,
							const PRECISION dt,
							const PRECISION delta,
							const PRECISION Sc,
							const PRECISION f,
							const PRECISION fp,
							const PRECISION dr,
							const unsigned int IEz, const unsigned int JEz,
							const unsigned int IHx, const unsigned int JHx,
							const unsigned int IHy, const unsigned int JHy,
							PRECISION *Ez_, PRECISION *Dz_, const PRECISION *EzMask_, const PRECISION *Hx_, const PRECISION *Bx_, const PRECISION *Hy_, const PRECISION *By_,
							const PRECISION *ax_, const PRECISION *ay_, const PRECISION *az_,
							PRECISION *PsiEzX_, PRECISION *PsiEzY_, const PRECISION *PsiHyX_, const PRECISION *PsiHxY_,
							const PRECISION kappex, const PRECISION kappey, const PRECISION kappmx, const PRECISION kappmy,
							const PRECISION bex, const PRECISION bey, const PRECISION bmx, const PRECISION bmy,
							const PRECISION Cex, const PRECISION Cey, const PRECISION Cmx, const PRECISION Cmy,
							PRECISION *Ezt, PRECISION *Eztt, PRECISION *Ezy1, PRECISION *Ezy2,
							const unsigned int x1, const unsigned int Y1, const unsigned int Y2,
							const unsigned int n,
							const unsigned int np,
							const unsigned int n0,
							const unsigned int nf)
{
	const unsigned int i = BlockX*blockIdx.x+threadIdx.x;
	const unsigned int j = BlockY*blockIdx.y+threadIdx.y;

	// ========================== Dz and Ez ==========================
	if (i>0)
	{
		// Psi arrays.
		if (j<JEz)
		{
			PsiEzX(i,j) = (Cex/delta)*(Hy(i,j,nf)-Hy(i-1,j,nf)) + bex*PsiEzX(i,j);
			PsiEzY(i,j) = (Cey/delta)*(-Hx(i,j+1,nf)+Hx(i,j,nf)) + bey*PsiEzY(i,j);
		}
		// Dz in normal space.
		if (j>PMLw-1 && j<JEz-PMLw)
		{
			Dz(i,j,nf) = Dz(i,j,n0) + (Hy(i,j,nf)-Hy(i-1,j,nf)-Hx(i,j+1,nf)+Hx(i,j,nf)) * dt/delta;
			Ez(i,j,nf) = EzMask(i,j) * (az(i,j,0)*(Dz(i,j,nf)-2.*Dz(i,j,n0)+Dz(i,j,np))+az(i,j,1)*(Dz(i,j,nf)-Dz(i,j,np))+az(i,j,2)*(2.*Ez(i,j,n0)-Ez(i,j,np))+az(i,j,3)*(2.*Ez(i,j,n0)+Ez(i,j,np))+az(i,j,4)*Ez(i,j,np));
		}
		// Dz in lower PML.
		if (j<PMLw)
		{
			Dz(i,j,nf) = Dz(i,j,n0) + dt*(((1./kappex)*(Hy(i,j,nf)-Hy(i-1,j,nf))+(1./kappey)*(-Hx(i,j+1,nf)+Hx(i,j,nf))) * 1./delta + PsiEzX(i,j) + PsiEzY(i,j));
			Ez(i,j,nf) = Dz(i,j,nf)/e0;
		}
		// Dz in upper PML.
		if (j>JEz-PMLw-1 && j<JEz)
		{
			Dz(i,j,nf) = Dz(i,j,n0) + dt*(((1./kappex)*(Hy(i,j,nf)-Hy(i-1,j,nf))+(1./kappey)*(-Hx(i,j+1,nf)+Hx(i,j,nf))) * 1./delta + PsiEzX(i,j) + PsiEzY(i,j));
			Ez(i,j,nf) = Dz(i,j,nf)/e0;
		}
	}
	else
	{
		// Psi arrays.
		if (j<JEz)
		{
			PsiEzX(0,j) = (Cex/delta)*(Hy(0,j,nf)-Hy(IEz-1,j,nf)) + bex*PsiEzX(0,j); // PBC
			PsiEzY(0,j) = (Cey/delta)*(-Hx(0,j+1,nf)+Hx(0,j,nf)) + bey*PsiEzY(0,j); // PBC
		}
		// Dz in normal space.
		if (j>PMLw-1 && j<JEz-PMLw)
		{
			Dz(0,j,nf) = Dz(0,j,n0) + (Hy(0,j,nf)-Hy(IEz-1,j,nf)-Hx(0,j+1,nf)+Hx(0,j,nf)) * dt/delta; // PBC
			Ez(0,j,nf) = EzMask(0,j) * (az(0,j,0)*(Dz(0,j,nf)-2.*Dz(0,j,n0)+Dz(0,j,np))+az(0,j,1)*(Dz(0,j,nf)-Dz(0,j,np))+az(0,j,2)*(2.*Ez(0,j,n0)-Ez(0,j,np))+az(0,j,4)*(2.*Ez(0,j,n0)+Ez(0,j,np))+az(0,j,4)*Ez(0,j,np)); // PBC
		}
		// Dz in lower PML.
		if (j<PMLw)
		{
			Dz(0,j,nf) = Dz(0,j,n0) + dt*(((1./kappex)*(Hy(0,j,nf)-Hy(IEz-1,j,nf))+(1./kappey)*(-Hx(0,j+1,nf)+Hx(0,j,nf))) * 1./delta + PsiEzX(0,j) + PsiEzY(0,j)); // PBC
			Ez(0,j,nf) = Dz(0,j,nf)/e0; // PBC
		}
		// Dz in upper PML.
		if (j>JEz-PMLw-1 && j<JEz)
		{
			Dz(0,j,nf) = Dz(0,j,n0) + dt*(((1./kappex)*(Hy(0,j,nf)-Hy(IEz-1,j,nf))+(1./kappey)*(-Hx(0,j+1,nf)+Hx(0,j,nf))) * 1./delta + PsiEzX(0,j) + PsiEzY(0,j)); // PBC
			Ez(0,j,nf) = Dz(0,j,nf)/e0; // PBC
		}
	}

	// ====================== Source ======================
	if (SourcePlane == 1 && i<IEz && j == SourceLocationY)
	{
		if (SourceChoice == 1)
			Ez(i,j,nf) += exp(-1.*pow(((PRECISION)n-(PRECISION)td)/((PRECISION)PulseWidth/4.),2)) * Sc;
		else if (SourceChoice == 2)
			Ez(i,j,nf) += sin(2.*pi*f*(PRECISION)n*dt) * Sc;
		else if (SourceChoice == 3)
			Ez(i,j,nf) += (1.-2.*pow(pi*fp*((PRECISION)n*dt-dr),2))*exp(-1.*pow(pi*fp*((PRECISION)n*dt-dr),2)) * Sc;

		Dz(i,j,nf) = e0*Ez(i,j,nf);
	}
	else if (i == SourceLocationX && j == SourceLocationY)
	{
		if (SourceChoice == 1)
			Ez(i,j,nf) += exp(-1.*pow(((PRECISION)n-(PRECISION)td)/((PRECISION)PulseWidth/4.),2)) * Sc;
		else if (SourceChoice == 2)
			Ez(i,j,nf) += sin(2.*pi*f*(PRECISION)n*dt) * Sc;
		else if (SourceChoice == 3)
			Ez(i,j,nf) += (1.-2.*pow(pi*fp*((PRECISION)n*dt-dr),2))*exp(-1.*pow(pi*fp*((PRECISION)n*dt-dr),2)) * Sc;

		Dz(i,j,nf) = e0*Ez(i,j,nf);
	}
	// Recording transmitted fields.
	if (i==IEz/2 && j==x1)
		Ezt[n] = Ez(i,j,nf);
	if (i==IEz/2 && j==2*J/3+PMLw+1)
		Eztt[n] = Ez(i,j,nf);
	// Fields for refractive index.
	if (i==IEz/2 && j==Y1)
		Ezy1[n] = Ez(i,j,nf);
	if (i==IEz/2 && j==Y2)
		Ezy2[n] = Ez(i,j,nf);
}